#include "hip/hip_runtime.h"

#include <iostream>
#include <math.h>
#include <vector>

#include "hip/hip_runtime.h"
#include ""

#include "gravitysim/kernel.hpp"

// function to add the elements of two arrays
__global__ void add(int n, float* x, float* y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int cudaAdd(void)
{
    int N = 1 << 20; // 1M elements

    float* x, * y;
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    // Run kernel on 1M elements on the CPU
    add <<<numBlocks, blockSize>>> (N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}

__global__ void runGravity(size_t objCount, Spaceobject* spaceobjects, Spaceobject* spaceobjectsStaticCopy, int* collisionList, double simDeltaTime) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < objCount; i+=stride) {
        Spaceobject& spobject = spaceobjects[i];
        for (size_t j = 0; j < objCount; j++) {
            if (i == j) continue;
            Spaceobject& otherSpobject = spaceobjectsStaticCopy[j];

            double xDist = otherSpobject.x - spobject.x;
            double yDist = otherSpobject.y - spobject.y;
            double distance = sqrt((xDist*xDist) + (yDist*yDist));
            double minDistance = spobject.radius + otherSpobject.radius;

            if (distance < minDistance) {
                // if self is not already designated for removal, designate other object for removal
                collisionList[i] = j;
                spobject.speedX = (1)*(((spobject.mass*spobject.speedX)+(otherSpobject.mass*otherSpobject.speedX))/(spobject.mass+otherSpobject.mass));
                spobject.speedY = (1)*(((spobject.mass*spobject.speedY)+(otherSpobject.mass*otherSpobject.speedY))/(spobject.mass+otherSpobject.mass));
                spobject.mass += otherSpobject.mass;
            } else {
                double unitDirX = (otherSpobject.x - spobject.x) / distance;
                double unitDirY = (otherSpobject.y - spobject.y) / distance;
                const double bigG = 6.674e-11;
                double force = bigG * ((spobject.mass*otherSpobject.mass)/(distance*distance));

                spobject.speedX += unitDirX * ((force / spobject.mass) * simDeltaTime);
                spobject.speedY += unitDirY * ((force / spobject.mass) * simDeltaTime);
            }
        }
        spobject.x += spobject.speedX * simDeltaTime;
        spobject.y += spobject.speedY * simDeltaTime;            
    }
}

void setupGravityMemory(size_t objectCount, Spaceobject** spaceobjects, Spaceobject** spaceobjectsStaticCopy, int** objectcollisions) {
    hipMallocManaged(spaceobjects, objectCount * sizeof(Spaceobject));
    hipMallocManaged(spaceobjectsStaticCopy, objectCount * sizeof(Spaceobject));
    hipMallocManaged(objectcollisions, objectCount * sizeof(int));
}

void freeGravityMemory(Spaceobject* spaceobjects, Spaceobject* spaceobjectsStaticCopy, int* objectcollisions) {
    hipFree(spaceobjects);
    hipFree(spaceobjectsStaticCopy);
    hipFree(objectcollisions);
}

void cudaGravity(std::vector<Spaceobject>& spaceobjectsRef, std::vector<int>& collisions, double simDeltaTime, Spaceobject* spaceobjects, Spaceobject* spaceobjectsStaticCopy, int* objectcollisions) {
    for (size_t i = 0; i < spaceobjectsRef.size(); i++) {
        spaceobjects[i] = spaceobjectsRef[i];
        spaceobjectsStaticCopy[i] = spaceobjectsRef[i];
        objectcollisions[i] = -1;
    }

    unsigned int blockSize = 256;
    unsigned int numBlocks = (static_cast<unsigned int>(spaceobjectsRef.size()) + blockSize - 1) / blockSize;

    runGravity <<<numBlocks, blockSize>>> (spaceobjectsRef.size(), spaceobjects, spaceobjectsStaticCopy, objectcollisions, simDeltaTime);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    for (size_t i = 0; i < spaceobjectsRef.size(); i++) {
        spaceobjectsRef[i] = spaceobjects[i];
        collisions[i] = objectcollisions[i];
    }
}